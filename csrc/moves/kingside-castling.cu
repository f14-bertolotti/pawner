#include "hip/hip_runtime.h"
#pragma once
#include <torch/extension.h>
#include "../chess-attacks.cu"
#include "../chess-consts.h"

__device__ bool kingside_castle(
    int env,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> players ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> boards  ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> actions
) {
    // performs kingside castling action
    // returns 0 if everything is ok
    // returns 1 if the action was a kingside castling but the conditions were not met
    
    const unsigned char player_king = players[env] * 6 + WHITE_KING;
    const unsigned char player_rook = players[env] * 6 + WHITE_ROOK;
    const unsigned char special = actions[env][4];
    const unsigned char castle_row  = players[env] == WHITE ? 7 : 0;
    const unsigned char king_source = castle_row * 8 + 4;
    const unsigned char rook_source = castle_row * 8 + 7;
    const unsigned char king_target = castle_row * 8 + 6;
    const unsigned char rook_target = castle_row * 8 + 5;

    const bool is_kingside_castle = (
        (actions[env][0] == 0   ) & // action source empty
        (actions[env][1] == 0   ) & // action source empty
        (actions[env][2] == 0   ) & // action target empty
        (actions[env][3] == 0   ) & // action target empty
        (special == KING_CASTLE )   // king castling action
    );

    const bool is_action_ok = ( 
        (boards[env][KING_MOVED + players[env]] == 0            ) & // king has not moved
        (boards[env][KINGSIDE_ROOK_MOVED + players[env]] == 0   ) & // king-side rook has not moved
        (boards[env][king_source] == player_king                ) & // king is in the right position
        (boards[env][rook_target] == EMPTY                      ) & // king-side is empty
        (boards[env][king_target] == EMPTY                      ) & // king-side is empty
        (boards[env][rook_source] == player_rook                ) & // king-side rook is in the right position
        (count_attacks(env, castle_row, 4, players, boards) == 0) & // king is not in check
        (count_attacks(env, castle_row, 5, players, boards) == 0) & // king-side 1 is not in check
        (count_attacks(env, castle_row, 6, players, boards) == 0)   // king-side 2 is not in check
    );

    boards[env][king_source] = (is_kingside_castle & is_action_ok) ? EMPTY       : boards[env][king_source];
    boards[env][rook_source] = (is_kingside_castle & is_action_ok) ? EMPTY       : boards[env][rook_source];
    boards[env][rook_target] = (is_kingside_castle & is_action_ok) ? player_rook : boards[env][rook_target];
    boards[env][king_target] = (is_kingside_castle & is_action_ok) ? player_king : boards[env][king_target];

    return is_kingside_castle & (!is_action_ok);
}

__global__ void kingside_castle_kernel(
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> boards  ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> actions ,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> players ,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> result
) {
    const int env = blockIdx.x * blockDim.x + threadIdx.x;
    if (env < boards.size(0)) result[env] = kingside_castle(env, players, boards, actions);
}
